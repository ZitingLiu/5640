#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#define BLOCK_SIZE 128
#define NUM_INPUT 33554432
#define NUM_CLASS 8
#define BIN 1000000/NUM_CLASS

__global__ void binning(int* arr,int* result,int* example){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid>=NUM_INPUT) return;
    
    
    int temp=arr[tid];
    for(int j=0;j<NUM_CLASS;j++){
        if(temp>=(j*BIN) && temp<(j+1)*BIN){
            atomicAdd(&result[j],1);
            if(example[j]==0){
                example[j]=temp;
            } 
            break;
        }
    }
    

}

int main(int argc, char const *argv[]){
    printf("Making histogram for %d inputs.\n",NUM_INPUT);
    int *arr, *result,*example;
    hipHostMalloc((void **) &arr, sizeof(int)*NUM_INPUT);
    hipHostMalloc((void **) &result, sizeof(int)*NUM_CLASS);
    hipHostMalloc((void **) &example, sizeof(int)*NUM_CLASS);

    srand(time(0));
    for(int i=0;i<NUM_INPUT;i++){
        arr[i]=rand()%1000000;
        //printf("%d ",arr[i]);
    }
    for(int i=0;i<NUM_CLASS;i++){
        result[i]=0;
        example[i]=0;
    }

    float gpu_elapsed_time_ms;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // start to count execution time of GPU version
    hipEventRecord(start, 0);

    int *d_arr, *d_result, *d_example;
    hipMalloc((void **) &d_arr, sizeof(int)*NUM_INPUT);
    hipMalloc((void **) &d_result, sizeof(int)*NUM_CLASS);
    hipMalloc((void **) &d_example, sizeof(int)*NUM_CLASS);

    hipMemcpy(d_arr, arr, sizeof(int)*NUM_INPUT, hipMemcpyHostToDevice);
    hipMemcpy(d_result, result, sizeof(int)*NUM_CLASS, hipMemcpyHostToDevice);
    hipMemcpy(d_example, example, sizeof(int)*NUM_CLASS, hipMemcpyHostToDevice);
    
    unsigned int grid_rows = (NUM_INPUT + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_rows);
    dim3 dimBlock(BLOCK_SIZE);

    binning<<<dimGrid,dimBlock>>>(d_arr,d_result,d_example);
    hipMemcpy(example, d_example, sizeof(int)*NUM_CLASS, hipMemcpyDeviceToHost);
    hipMemcpy(result, d_result, sizeof(int)*NUM_CLASS, hipMemcpyDeviceToHost);
    
    hipDeviceSynchronize();
    hipDeviceSynchronize();
    // time counting terminate
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop);
    printf("Elapsed time: %f ms.\n\n", gpu_elapsed_time_ms);
    
    printf("Number of elements in each of the %d bins are: \n",NUM_CLASS);
    for(int i=0;i<NUM_CLASS;i++){
        printf("%d ",result[i]);
    }
    printf("\n");

    printf("One number from each of the %d bins to show that result is correct \n",NUM_CLASS);
    for(int i=0;i<NUM_CLASS;i++){
        printf("%d ",example[i]);
    }
    printf("\n");

    return 0;
}