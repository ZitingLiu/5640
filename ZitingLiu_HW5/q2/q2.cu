#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#define BLOCK_SIZE 16
#define n 64

__global__ void gpu_nn(float *a, float *b){
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    //printf("Hello, world from block %d,%d\n", col, row);
    if(col>=1 && row>=1 && col<n-1 && row<n-1){
        for(int k=1;k<n-1;k++){
            a[row*n*n+col*n+k]=0.8*(b[(row-1)*n*n+col*n+k]+b[(row+1)*n*n+col*n+k]
            +b[row*n+(col+1)*n+k]+b[row*n+(col-1)*n+k]+b[row*n+col*n+k+1]+b[row*n+col*n+k]-1);
            //a[row*n*n+col*n+k]=1;
        }
    }
}




int main(int argc, char const *argv[]){
    srand(time(0));
    float a[n*n*n]={0};
    float b[n*n*n]={0};



    float *h_a,*h_b;
    hipHostMalloc((void **) &h_a, sizeof(float)*n*n*n);
    hipHostMalloc((void **) &h_b, sizeof(float)*n*n*n);

    for(int i=0;i<n;i++){
        for(int j=0;j<n;j++){
            for(int k=0;k<n;k++){
                float temp=(float)(rand()%10)+1;
                //printf("%f ",temp);
                
                h_b[i*n*n+j*n+k]=temp;
                b[i*n*n+j*n+k]=temp;
                h_a[i*n*n+j*n+k]=0.0;
            }
        }
    }
    printf("\n");
    float gpu_elapsed_time_ms;

    // some events to count the execution time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // start to count execution time of GPU version
    hipEventRecord(start, 0);

    float *d_a,*d_b;
    hipMalloc((void **) &d_a, sizeof(float)*n*n*n);
    hipMalloc((void **) &d_b, sizeof(float)*n*n*n);

    hipMemcpy(d_a,h_a, sizeof(float)*n*n*n,hipMemcpyHostToDevice);
    hipMemcpy(d_b,h_b, sizeof(float)*n*n*n,hipMemcpyHostToDevice);

    unsigned int grid = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid,grid);
    dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);

    gpu_nn<<<dimGrid,dimBlock>>>(d_a,d_b);
    
    hipMemcpy(h_a, d_a, sizeof(float)*n*n*n, hipMemcpyDeviceToHost);
    hipMemcpy(h_b, d_b, sizeof(float)*n*n*n, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    // time counting terminate
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop);
    printf("Elapsed time %f", gpu_elapsed_time_ms);
    printf("\n\n");
    
    
    ///////////////////////cpu version just to verify result
    for(int i=1;i<n-1;i++){
        for(int j=1;j<n-1;j++){
            for(int k=1;k<n-1;k++){
                a[i*n*n+j*n+k]=0.8*(b[(i-1)*n*n+j*n+k]+b[(i+1)*n*n+j*n+k]
            +b[i*n+(j+1)*n+k]+b[i*n+(j-1)*n+k]+b[i*n+j*n+k+1]+b[i*n+j*n+k]-1);
            }
        }
    }
    bool all_ok=true;
    int ind=0;
    for(int i=1;i<n-1;i++){
        for(int j=1;j<n-1;j++){
            for(int k=1;k<n-1;k++){
                if(h_a[i*n*n+j*n+k]!=a[i*n*n+j*n+k]){
                    all_ok=0;
                    ind=i*n*n+j*n+k;
                    break;
                } 
            }
        }
    }
    if(all_ok){
        printf("All results are correct!\n");
    }else{
        printf("something went wrong at index %d.\n",ind);
    }
    
    printf("CUDA     CPU\n");
    
    for(int i=1;i<n-1;i++){
        for(int j=1;j<n-1;j++){
            for(int k=1;k<n-1;k++){
                
                printf("%6.3f ",h_a[i*n*n+j*n+k]);
                printf("%6.3f ",a[i*n*n+j*n+k]);
                printf("\n");
            }
        }
    }

    printf("\n\n");

}

