#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#define BLOCK_SIZE 16
#define N 10000

__global__ void gpu_dart(int* a,int* result){
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if(row<N && col<N){
        //printf("%d\n",a[N*row+col]);
        if(a[N*row+col]<314) result[N*row+col]=1;
    }

}




int main(int argc, char const *argv[]){
    int *h_a,*h_total;
    hipHostMalloc((void **) &h_a, sizeof(int)*N*N);
    hipHostMalloc((void **) &h_total, sizeof(int)*N*N);
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            int temp= rand() % 400;
            h_a[i * N + j]=temp; 
            h_total[i * N + j]=0; 
            
        }
    }

    int *d_a, *d_total;
    hipMalloc((void **) &d_a, sizeof(int)*N*N);
    hipMalloc((void **) &d_total, sizeof(int)*N*N);

    hipMemcpy(d_a, h_a, sizeof(int)*N*N, hipMemcpyHostToDevice);
    hipMemcpy(d_total, h_total, sizeof(int)*N*N, hipMemcpyHostToDevice);

    unsigned int grid = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid, grid);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    gpu_dart<<<dimGrid,dimBlock>>>(d_a,d_total);

    hipMemcpy(h_total, d_total, sizeof(int)*N*N, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    int sum=0;
    for(int i=0;i<N*N;i++) {
        if(h_total[i]==1) sum++;
    }
    double pi=(double)sum/(double)(N*N)*4.0;
    printf("Estimated result with %d inputs : %f\n",N*N,pi);


}